#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#include ""
#endif // !__HIPCC__

#include "hip/hip_vector_types.h"
#include "vector_functions.h"
#include "hip/hip_runtime.h"
#include "default.h"

enum
{
	BlockSize = 512,
};

__global__
void renderKernel(size_t count, size_t width, size_t height, size_t samples, uchar4 *__cuda__pRenderBuffer)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  int x = i % width;
  int y = i / width;

  if (i >= count)
    return;

  __cuda__pRenderBuffer[i] = make_uchar4((uint8_t)(x / (float)width * 0xFF) , (uint8_t)(y / (float)height * 0xFF), (uint8_t)((x + y) / (float)(width + height) * 0xFF), 0);
}

extern "C" 
{
  void Init(size_t width, size_t height, uchar4 **p__cuda__pRenderBuffer)
  {
    hipMalloc(p__cuda__pRenderBuffer, sizeof(uchar4) * width * height);
  }

	void Render(size_t width, size_t height, size_t samples, uchar4 *__cuda__pRenderBuffer)
	{
    renderKernel<<<width * height, BlockSize>>>(width * height, width, height, samples, __cuda__pRenderBuffer);

    hipDeviceSynchronize();
	}

  void Cleanup(uchar4 **p__cuda__pRenderBuffer)
  {
    hipFree(*p__cuda__pRenderBuffer);
  }
}